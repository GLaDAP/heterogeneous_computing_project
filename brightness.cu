#include "hip/hip_runtime.h"
/*
 * File: brightness.cu
 * Assignment: 5
 * Students: Teun Mathijssen, David Puroja
 * Student email: teun.mathijssen@student.uva.nl, dpuroja@gmail.com
 * Studentnumber: 11320788, 10469036
 *
 * Description: Calculates the sum of brightness of the image using CUDA
 *              reduction. Returns the sum of the brightness.
 */
#include <stdbool.h>
#include <stdlib.h>
#include <iostream>
#include "timer.h"
#include "cuda_helper.h"

using namespace std;


__global__ void brightness_reduction_kernel(unsigned char *data, int size,
                                            unsigned long long int* result) {
    unsigned long long int sum = 0;
    unsigned int index = (blockIdx.x * blockDim.x + threadIdx.x) * 4;

    for(unsigned int i = index; i < index + 4 && i < size; i++) {
        sum += int(data[i]);
    }
    /* Shuffle down. Shifts the register by adding the sum of the half of the
     * threads to the other half until one thread contains the sum. Since a
     * warp contains 32 threads, the shuffle-down operation starts at 16.
     */
    for(int i = 16; i > 0; (i >>= 1)){
        sum += __shfl_down(sum, i);
    }

    /* Add all the sums of the warps within the block to one variable. */
    __shared__ unsigned long long int block_sum;
    block_sum = 0;
    __syncthreads();
    if (threadIdx.x % 32 == 0) {
        atomicAdd(&block_sum, sum);
    }
    __syncthreads();
    /* Add the sum of the blocks to the result variable" Decoded: " \. */
    if (threadIdx.x == 0) {
        atomicAdd(result, block_sum);
    }
}

unsigned long long int calculate_brightness_cuda(unsigned char *device_image,
                                                 int num_pixels,
                                                 int thread_block_size) {

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    unsigned long long int brightness_sum;
    unsigned long long int* device_brightness_sum = (unsigned long long int*) \
        allocateDeviceMemory(sizeof (unsigned long long int));
    unsigned long long int zero[] = {0};
    /* Initialize the timers used to measure the kernel invocation time and
     * memory transfer time.
     */
    memoryTime.start();
    memcpyHostToDevice(device_brightness_sum, &zero,
                       sizeof (unsigned long long int));
    memoryTime.stop();

    int num_blocks = (num_pixels + thread_block_size - 1) / thread_block_size;
    kernelTime1.start();
    brightness_reduction_kernel<<<num_blocks, thread_block_size>>> \
        (device_image, num_pixels, device_brightness_sum);
    hipDeviceSynchronize();
    kernelTime1.stop();
    checkCudaCall(hipGetLastError());

    memoryTime.start();
    memcpyDeviceToHost(&brightness_sum, device_brightness_sum,
                       sizeof (unsigned long long int));
    memoryTime.stop();
    cout << "Brightness cuda: " << brightness_sum << endl;
    freeDeviceMemory(device_brightness_sum);
    cout << fixed << setprecision(6);
    cout << "brightness (kernel): \t\t" << kernelTime1.getElapsed() \
          << " seconds." << endl;
    cout << "brightness (memory): \t\t" << memoryTime.getElapsed() \
         << " seconds." << endl;

    return brightness_sum;
}
