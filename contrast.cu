#include "hip/hip_runtime.h"
/*
 * File: contrast.cu
 * Assignment: 5
 * Students: Teun Mathijssen, David Puroja
 * Student email: teun.mathijssen@student.uva.nl, david.puroja@student.uva.nl
 * Studentnumber: 11320788, 10469036
 *
 * Description: Applies the contrast filter on the image using CUDA.
 */

#include <stdbool.h>
#include <stdlib.h>
#include <iostream>
#include "timer.h"
#include "cuda_helper.h"

using namespace std;

/* The maximum value we can use as RGB component. */
#define RGB_MAX_VALUE 255

/* CUDA contrast filter kernel. Calculates for each pixel in its range the
 * new value using the mean and denominator of the brightness sum. This kernel
 * executes in two steps: the first step are the threads with a pixel which
 * satisfies the requirement of the second if-statement. The second stage is the
 * else. This is due the architecture of the GPU where all the threads in a warp
 * executes the same instruction.
 */
__global__ void filter_contrast_kernel(unsigned char *image_data, int size,
                                       double mean, double denominator) {
    unsigned int index = (blockIdx.x * blockDim.x + threadIdx.x);
    if (index < size) {
        if (image_data[index] >= mean) {
            image_data[index] = (sqrt(image_data[index] - mean)
                              / denominator * RGB_MAX_VALUE);
        }
        else {
            image_data[index] = 0;
        }
    }
}

/* Allocates resources on the GPU, times the memory and kernel operations,
 * and executes the kernel.
 */
void filter_contrast_cuda(unsigned char *image_data, int num_pixels,
                          long brightness_sum, int max_index,
                          int thread_block_size) {
    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    /* Copy the image to the device. */
    unsigned char* device_image = (unsigned char*) allocateDeviceMemory( \
        num_pixels * sizeof (unsigned char));
    memcpyHostToDevice(device_image, image_data, \
                       num_pixels * sizeof (unsigned char));

    double brightness_mean = (double) (brightness_sum / (double) num_pixels);
    double denominator = sqrt(RGB_MAX_VALUE - brightness_mean);

    int num_blocks = (max_index + thread_block_size - 1) / thread_block_size;

    kernelTime1.start();
    filter_contrast_kernel<<<num_blocks, thread_block_size>>> \
        (device_image, max_index, brightness_mean, denominator);
        hipDeviceSynchronize();
    kernelTime1.stop();
    checkCudaCall(hipGetLastError());

    /* Copy the result image back to the GPU. */
    memoryTime.start();
    memcpyDeviceToHost(image_data, device_image, \
                       num_pixels * sizeof (unsigned char));
    memoryTime.stop();

    /* Free used memory on the GPU. */
    freeDeviceMemory(device_image);

    /* Print the elapsed time. */
    cout << fixed << setprecision(6);
    cout << "contrast (kernel): \t\t" << kernelTime1.getElapsed() \
          << " seconds." << endl;
    cout << "contrast (memory): \t\t" << memoryTime.getElapsed() \
         << " seconds." << endl;
}
