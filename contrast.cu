#include "hip/hip_runtime.h"
/*
 * File: main.cpp
 * Assignment: 5
 * Students: Teun Mathijssen, David Puroja
 * Student email: teun.mathijssen@student.uva.nl, dpuroja@gmail.com
 * Studentnumber: 11320788, 10469036
 *
 * Description: This file contains sequential implementations of different
 * image processing functions.
 */

#include <stdbool.h>
#include <stdlib.h>
#include <iostream>
#include "timer.h"
#include "cuda_helper.h"
#include "brightness.h"
using namespace std;

/* The maximum value we can use as RGB component. */
#define RGB_MAX_VALUE 255

/* Compare with a OpenMP implementation? */
__global__ void filter_contrast_kernel(unsigned char *image_data,
                                       int size, float mean,
                                       float denominator) {
    unsigned int index = (blockIdx.x * blockDim.x + threadIdx.x);
    if(index < size) {
        if(image_data[index] >= mean) {
            image_data[index] = (sqrt(image_data[index] - mean)
                              / denominator * RGB_MAX_VALUE);
        }
        else {
            image_data[index] = 0;
        }
    }
}

void filter_contrast_cuda(unsigned char *image_data, int num_pixels, int max_index) {
    int thread_block_size = 512;
    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    unsigned char* device_image = (unsigned char*) allocateDeviceMemory( \
        num_pixels * sizeof (unsigned char));
    memcpyHostToDevice(device_image, image_data, num_pixels * sizeof(unsigned char));

    int brightness_sum = calculate_brightness_cuda(device_image, num_pixels);
    int num_blocks = (num_pixels + thread_block_size - 1) / thread_block_size;

    /* And now the contrast */
    float brightness_mean = (double) brightness_sum / (double) num_pixels;
    float denominator = sqrt(RGB_MAX_VALUE - brightness_mean);
    kernelTime1.start();
    filter_contrast_kernel<<<num_blocks, thread_block_size>>> \
        (device_image, max_index, brightness_mean, denominator);
        hipDeviceSynchronize();

    kernelTime1.stop();
    checkCudaCall(hipGetLastError());

    /* Copy the result image back to the GPU. */
    memoryTime.start();
    memcpyDeviceToHost(image_data, device_image, num_pixels * sizeof (unsigned char));
    memoryTime.stop();

    /* Free used memory on the GPU. */
    freeDeviceMemory(device_image);
    cout << fixed << setprecision(6);
    cout << "contrast (kernel): \t\t" << kernelTime1.getElapsed() \
          << " seconds." << endl;
    cout << "contrast (memory): \t\t" << memoryTime.getElapsed() \
         << " seconds." << endl;
}
